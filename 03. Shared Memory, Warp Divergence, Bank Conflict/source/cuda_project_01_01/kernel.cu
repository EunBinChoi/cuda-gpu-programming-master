
// reduce 0 프로그램 수정 
// reduce 0 => 공유메모리를 사용하지 않는 버전
// reduce 1 => 공유메모리 사용버전
// reduce 2 => 워프 분기 해결
// reduce 3 => 뱅크 충돌 해결

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void reduce(int*a, int*o, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	for(int s = 1; s < blockDim.x; s *= 2){ // 한 블럭 내의 차수까지 반복
		if(idx % (2*s) == 0)
		{
			// 옆에 있는 값과 비교하여 최소값을 저장
			if(a[idx] < a[idx + s])  
				a[idx] = a[idx];
			else
				a[idx] = a[idx + s];
		} 
		__syncthreads();
	}

	if(threadIdx.x == 0) o[blockIdx.x] = a[idx];
}
int main(){
	const int N		= 1000 * 1024;
	int	block_size	= 1024;
	int	block_num	= (N + block_size -1) / block_size;

	int *a, *o;
	a = (int *)malloc(sizeof(int) * N);
	o = (int *)malloc(sizeof(int) * block_num);
	
	for(int i = 0 ; i < N ; i++)
	{
		a[i] = (i % 2 == 0)? i : -i;
	}

	int *id, *od;
	hipMalloc((void **)&id, sizeof(int)*N);
	hipMalloc((void **)&od, sizeof(int)*block_num);
	
	hipEvent_t start, end;
	float gpu_time;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipMemcpy(id,a,sizeof(int)*N, hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	reduce<<<block_num, block_size>>>(id,od,N);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time, start, end);
	
	printf("\n");
	printf("gpu_time = %f\n", gpu_time);

	hipMemcpy(o,od,sizeof(int)*block_num, hipMemcpyDeviceToHost);

	// block 당 나온 최소값들 중에서 최소값을 구함
	int min = INT_MAX;
	for(int i = 0 ; i < block_num ; i ++){
		if(o[i] < min) min = o[i];
	}

	printf("\nGPU 최소값 = %d\n", min);
	

	// ---------------------------------------
	// CPU 에서 최소값
	int min_cpu = INT_MAX;
	for(int i = 0 ; i < N ; i ++)
	{
		if(a[i] < min_cpu) 
			min_cpu = a[i];
	}
	
	printf("CPU 최소값 = %d\n", min_cpu);
	
	free(a); free(o);
	hipFree(id); hipFree(od);

	return 0;
}