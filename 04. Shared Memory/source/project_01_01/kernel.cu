// 짝수 홀수 생각해보기
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#define SHARED 1024

__global__ void reduce(int*a ,int *o, int n)
{
	__shared__ int sa[SHARED]; // 공유메모리

	int tid = threadIdx.x;
	int idx = blockIdx.x * (blockDim.x*2) + threadIdx.x;

	// 교수님이 말한 예외처리 하다가 맘
	/*if(idx > n){
		if(threadIdx.x < SHARED)
		{
			sa[tid] = 0;
			return;
		}
	}*/
	
	if(idx + blockDim.x >= n) sa[tid] = a[idx];
	else sa[tid] = a[idx] + a[idx + blockDim.x];
	
	//if(idx < n) sa[tid] = a[idx] + a[idx + blockDim.x];
	
	__syncthreads();

	for(int s = blockDim.x/2; s >= 1; s >>= 1){
		if(tid < s)
			sa[tid] = sa[tid] + sa[tid + s];
		__syncthreads();
	}
	
	if(tid == 0) o[blockIdx.x] = sa[tid];
}

int main(){
	const int N = 1024*1000;
	int block_size = 1024;
	int block_num = ((N + block_size -1)/block_size);
	
	int *a, *o;
	a = (int *)malloc(sizeof(int) * N);
	o = (int *)malloc(sizeof(int) * block_num);
	for(int i = 0 ; i < N; i++) {
		a[i] = (i % 2 ==0) ? i : -i;
		//printf("%d ", a[i]);
	}

	int *id, *od;
	hipMalloc((void **)&id, sizeof(int)*N);
	hipMalloc((void **)&od, sizeof(int)*block_num);
	
	hipEvent_t start, end;
	float gpu_time;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipMemcpy(id,a,sizeof(int)*N, hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	reduce<<<block_num/2, block_size>>>(id,od,N);
	
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time, start, end);
	
	printf("\n>>gpu_time = %f\n", gpu_time);
	//printf("%d\n\n", N/2);
	hipMemcpy(o,od,sizeof(int)*block_num, hipMemcpyDeviceToHost);
	int sum = 0;
	for(int i = 0 ; i < block_num; i++){
		//printf("%d ", o[i]);
		sum += o[i];
	}
	printf("\n");
	printf("GPU : %d", sum);
	printf("\n");

	//CPU
	int sum2 = 0;
	for(int i = 0 ; i < N ; i ++)
	{
		sum2 += a[i];
	}
	
	printf("CPU : %d\n",sum2);

	hipError_t err;
	err = hipGetLastError();
	printf("%s\n", hipGetErrorString(err));

	free(a); free(o);
	hipFree(id); hipFree(od);

	return 0;
}